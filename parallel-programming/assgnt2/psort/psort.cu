#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define NUM 1048576 
#define NUM_THREADS 512
#define NUM_BLOCKS 2048

/* Function to sort threads in each block using merge sort */

__global__ void sort_blocks(int *a)
{
  int i=2;

  __shared__ int temp [NUM_THREADS];

  while (i <= NUM_THREADS)
  {
    if ((threadIdx.x % i)==0)
    {
      int begin1 = threadIdx.x + (blockIdx.x * blockDim.x);
      int end1 = begin1 + i/2;
      int begin2 = end1;
      int end2 = begin2 + i/2;
      int target = threadIdx.x;

      do
      {
         if ((begin1 == end1) && (begin2 < end2))
           temp[target++] = a[begin2++];
        
         else if ((begin2 == end2) && (begin1 < end1))
           temp[target++] = a[begin1++];
        
         else if (a[begin1] < a[begin2])
           temp[target++] = a[begin1++];

         else
           temp[target++] = a[begin2++];

       }
       while ((begin1!=end1) && (begin2!=end2));
     } 

     __syncthreads();

     a[threadIdx.x + (blockIdx.x*blockDim.x)] = temp[threadIdx.x];

     __syncthreads();

     i *= 2;

   }
} 

/* Function to merge the sorted blocks using merge sort */

__global__ void merge_blocks(int *a, int *temp, int sortedsize)
{
  int id = blockIdx.x; 
  int begin1 = id * 2 * sortedsize;      
  int end1 = begin1 + sortedsize;
  int begin2 = end1;
  int end2 = begin2 + sortedsize;
  int target = id * 2 * sortedsize;

  do
  {
    if ((begin1 == end1) && (begin2 < end2))
      temp[target++] = a[begin2++];

    else if ((begin2 == end2) && (begin1 < end1))
      temp[target++] = a[begin1++];

    else if (a[begin1] < a[begin2])
      temp[target++] = a[begin1++];

    else
      temp[target++] = a[begin2++];

  }
  while ((begin1!=end1) && (begin2!=end2));

} 

int main()
{
  int *a = (int *) malloc (NUM * sizeof (int));
  int *dev_a, *dev_temp; 

  hipMalloc((void **) &dev_a, NUM*sizeof(int));
  hipMalloc((void **) &dev_temp, NUM*sizeof(int)); 
  
  for (int i = 0; i < NUM; i++)
  {
    a[i] = rand () % 10000; 
  }

  /* timing */
  hipEvent_t start, stop;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  /* timing */
 
  hipMemcpy(dev_a, a, NUM*sizeof(int), hipMemcpyHostToDevice); 
  
  /* Sort the elements corresponding to the threads in each block */

  sort_blocks<<<NUM_BLOCKS, NUM_THREADS>>>(dev_a);

  hipMemcpy(a, dev_a, NUM*sizeof(int), hipMemcpyDeviceToHost); 

  /* Merge the sorted blocks */
		
  int blocks = NUM_BLOCKS/2;
  int sortedsize = NUM_THREADS;

  while (blocks > 0)
  {
     merge_blocks<<<blocks, 1>>>(dev_a, dev_temp, sortedsize);
     hipMemcpy (dev_a, dev_temp, NUM*sizeof(int), hipMemcpyDeviceToDevice);
     
     blocks /= 2;
     sortedsize *= 2;
  }

  hipMemcpy(a, dev_a, NUM*sizeof(int), hipMemcpyDeviceToHost);

  /* timing */
  hipEventRecord (stop, 0);
  hipEventSynchronize (stop);
  hipEventElapsedTime (&time, start, stop);
  hipEventDestroy (start);
  hipEventDestroy (stop); 
  /* timing */
		
  bool passed = true;
   
  for(int i = 1; i < NUM; i++)
  {
    if (a [i-1] > a [i])
      passed = false;
  }
  
  printf("\nTest %s\n", passed ? "PASSED" : "FAILED");
  printf("Time : %f\n", time);
	
  hipDeviceReset();
		
  return 0;
}

