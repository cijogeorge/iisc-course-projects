#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define NUM 4096
#define NUM_THREADS 512
#define NUM_BLOCKS 8

/* Function to implement c[i] = a[i] + b[i] non coalesced memory access */

__global__ void no_coalesce(int *a, int *b, int *c)
{
  int idx = threadIdx.x;
  idx = NUM_THREADS - idx - 1;
  int index = idx + (blockIdx.x * blockDim.x);
  c [index] = a [index] + b [index]; 
}

/* Function to implement c[i] = a[i] + b[i], coalesced memory access */

__global__ void coalesce (int *a, int *b, int *c)
{
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  c [index] = a [index] + b [index];
}

int main ()
{
  /* Timing */
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop); 
  /* Timing */

  int a[NUM], b[NUM], c[NUM], i;
  float time1, time2;
  bool passed;

  for (i=0; i<NUM; i++)
  {
    a[i] = i;
    b[i] = i;
  }
    
  int *dev_a, *dev_b, *dev_c;
    
  hipMalloc(&dev_a, NUM * sizeof(int));
  hipMemcpy(dev_a, a, NUM * sizeof(int), hipMemcpyHostToDevice);
  hipMalloc(&dev_b, NUM * sizeof(int));
  hipMemcpy(dev_b, b, NUM * sizeof(int), hipMemcpyHostToDevice);
  hipMalloc(&dev_c, NUM * sizeof(int));
  hipMemcpy(dev_c, c, NUM * sizeof(int), hipMemcpyHostToDevice);
   
  /* Timing */
  hipEventRecord(start, 0); 
  /* Timing */

  no_coalesce<<<NUM_BLOCKS, NUM_THREADS>>>(dev_a, dev_b, dev_c);
  hipDeviceSynchronize();

  /* Timing */
  hipEventRecord (stop, 0); 
  hipEventSynchronize (stop); 
  hipEventElapsedTime (&time1, start, stop); 
  /* Timing */

  hipMemcpy(c, dev_c, NUM*sizeof(int), hipMemcpyDeviceToHost);

  passed = true;

  for (i=0; i<NUM; i++)
  { 
    if (c [i] != a [i] + b [i])
      passed = false;
  }
  
  printf ("\nNon-Coalesced:\t%s\nTime:\t%f\n", passed ? "PASSED" : "FAILED", time1);

  /* Timing */
  hipEventRecord(start, 0);
  /* Timing */

  coalesce<<<NUM_BLOCKS, NUM_THREADS>>>(dev_a, dev_b, dev_c);
  hipDeviceSynchronize();

  /* Timing */
  hipEventRecord (stop, 0);   
  hipEventSynchronize (stop); 
  hipEventElapsedTime (&time2, start, stop);
  /* Timing */

  hipMemcpy(c, dev_c, NUM*sizeof(int), hipMemcpyDeviceToHost);
   
  passed = true;

  for (i=0; i<NUM; i++)
  { 
    if (c [i] != a [i] + b [i])
      passed = false;
  }
  
  printf ("\nCoalesced:\t%s\nTime:\t%f\n", passed ? "PASSED" : "FAILED", time2);
  
  /* Timing */
  hipEventDestroy (start);  
  hipEventDestroy (stop);
  /* Timing */
        
  hipFree (dev_a);
  hipFree (dev_b);
  hipFree (dev_c);

  return 0;
}
